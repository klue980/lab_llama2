// Author: Wonseok Lee (aram_fahter@naver.com)

// Last update: 2021-03-09(TUE)

#include <iostream>
#include <cstdlib>

int main(int argc, char** argv)
{
  hipError_t error;

  int number_of_devices;
  error = hipGetDeviceCount(&number_of_devices);
  if (error)
  {
    std::cout << hipGetErrorString(error) << std::endl;
    exit(-1);
  }

  for (int dev_idx = 0; dev_idx < number_of_devices; ++dev_idx)
  {
    hipDeviceProp_t props;
    error = hipGetDeviceProperties(&props, dev_idx);
    if (error)
    {
      std::cout << hipGetErrorString(error) << std::endl;
      exit(-1);
    }

    printf("Device Index: %d\n", dev_idx);
    printf(
        "  - name                       : %s\n"
        "  - totalGlobalMem             : %zu bytes\n"
        "  - sharedMemPerBlock          : %zu bytes\n"
        "  - regsPerBlock               : %d\n"
        "  - warpSize                   : %d\n"
        "  - memPitch                   : %zu bytes\n"
        "  - maxThreadsPerBlock         : %d\n"
        "  - maxThreadsDim              : %d x %d x %d\n"
        "  - maxGridSize                : %d x %d x %d\n"
        "  - clockRate                  : %d KHz\n"
        "  - totalConstMem              : %zu bytes\n"
        "  - major                      : %d\n"
        "  - minor                      : %d\n"
        "  - textureAlignment           : %zu\n"
        "  - deviceOverlap              : %d(1 for True, 0 for False)\n"
        "  - multiProcessorCount        : %d\n"
        "  - kernelExecTimeoutEnabled   : %d(1 for True, 0 for False)\n"
        "  - integrated                 : %d(1 for True, 0 for False)\n"
        "  - canMapHostMemory           : %d(1 for True, 0 for False)\n"
        "  - computeMode                : %d(0 for Default, 1 for Exclusive, 2 for Prohibited, 3 for ExclusiveProcess)\n"
        "  - maxTexture1D               : %d\n"
        "  - maxTexture2D               : %d x %d\n"
        "  - maxTexture3D               : %d x %d x %d\n"
        "  - maxTexture1DLayered        : %d x %d\n"
        /*"  - maxTexture2DLayered        : %d x %d x %d\n"*/
        "  - surfaceAlignment           : %zu\n"
        "  - concurrentKernels          : %d(1 for True, 0 for False)\n"
        "  - ECCEnabled                 : %d(1 for True, 0 for False)\n"
        "  - pciBusID                   : %08X\n"
        "  - pciDeviceID                : %08X\n"
        "  - pciDomainID                : %08X\n"
        "  - tccDriver                  : %d(1 for True, 0 for False)\n"
        "  - asyncEngineCount           : %d\n"
        "  - unifiedAddressing          : %d(1 for True, 0 for False)\n"
        "  - memoryClockRate            : %d KHz\n"
        "  - memoryBusWidth             : %d bits\n"
        "  - l2CacheSize                : %d bytes\n"
        "  - maxThreadsPerMultiProcessor: %d\n",
        props.name,
        props.totalGlobalMem,
        props.sharedMemPerBlock,
        props.regsPerBlock,
        props.warpSize,
        props.memPitch,
        props.maxThreadsPerBlock,
        props.maxThreadsDim[0], props.maxThreadsDim[1], props.maxThreadsDim[2],
        props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2],
        props.clockRate,
        props.totalConstMem,
        props.major,
        props.minor,
        props.textureAlignment,
        props.deviceOverlap,
        props.multiProcessorCount,
        props.kernelExecTimeoutEnabled,
        props.integrated,
        props.canMapHostMemory,
        props.computeMode,
        props.maxTexture1D,
        props.maxTexture2D[0], props.maxTexture2D[1],
        props.maxTexture3D[0], props.maxTexture3D[1], props.maxTexture3D[2],
        props.maxTexture1DLayered[0], props.maxTexture1DLayered[1],
        /*props.maxTexture2DLayered[0], props.maxTexture2DLayered[1], maxTexture2DLayered[2],*/
        props.surfaceAlignment,
        props.concurrentKernels,
        props.ECCEnabled,
        props.pciBusID,
        props.pciDeviceID,
        props.pciDomainID,
        props.tccDriver,
        props.asyncEngineCount,
        props.unifiedAddressing,
        props.memoryClockRate,
        props.memoryBusWidth,
        props.l2CacheSize,
        props.maxThreadsPerMultiProcessor);
  }

  return 0;
}